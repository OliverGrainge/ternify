#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>

// CUDA Kernel for computing batched matrix multiplication (naive_matmul)
__global__ void matmul_kernel(const float* A, const float* B, float* C, int64_t BATCH, int64_t M, int64_t N, int64_t K) {
    int batch_idx = blockIdx.z;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (batch_idx < BATCH && row < M && col < K) {
        const float* A_batch = A + batch_idx * M * N;
        const float* B_batch = B;
        float* C_batch = C + batch_idx * M * K;

        float sum = 0.0f;
        for (int64_t k = 0; k < N; ++k) {
            sum += A_batch[row * N + k] * B_batch[k * K + col];
        }
        C_batch[row * K + col] = sum;
    }
}

torch::Tensor matmul_gpu(torch::Tensor A, torch::Tensor B) {
    // Ensure the tensors are on CUDA
    TORCH_CHECK(A.device().is_cuda(), "Tensor 'A' must be on CUDA");
    TORCH_CHECK(B.device().is_cuda(), "Tensor 'B' must be on CUDA");

    int64_t BATCH, M, N, K;
    torch::Tensor C;

    if (A.dim() == 3) {
        // Batched case
        BATCH = A.size(0);
        M = A.size(1);
        N = A.size(2);
        K = B.size(1);
        TORCH_CHECK(B.size(0) == N, "Tensor dimensions are not compatible for matrix multiplication");

        // Create an output tensor
        C = torch::zeros({BATCH, M, K}, torch::dtype(A.dtype()).device(torch::kCUDA));
    } else if (A.dim() == 2) {
        // Non-batched case
        BATCH = 1;
        M = A.size(0);
        N = A.size(1);
        K = B.size(1);
        TORCH_CHECK(B.size(0) == N, "Tensor dimensions are not compatible for matrix multiplication");

        // Create an output tensor
        C = torch::zeros({M, K}, torch::dtype(A.dtype()).device(torch::kCUDA));
    } else {
        TORCH_CHECK(false, "Tensor 'A' must be either 2-dimensional or 3-dimensional");
    }

    // Define block and grid sizes
    dim3 block_size(16, 16);
    dim3 grid_size((M + block_size.x - 1) / block_size.x, (K + block_size.y - 1) / block_size.y, BATCH);

    // Get pointers to the underlying data
    const float* A_ptr = A.data_ptr<float>();
    const float* B_ptr = B.data_ptr<float>();
    float* C_ptr = C.data_ptr<float>();

    // Launch the CUDA kernel
    matmul_kernel<<<grid_size, block_size>>>(A_ptr, B_ptr, C_ptr, BATCH, M, N, K);
    hipDeviceSynchronize();

    return C;
}

// Use PyBind11 to bind the function to Python
PYBIND11_MODULE(functional, m) {
    m.def("matmul_gpu", &matmul_gpu, "Naive Matrix Multiplication (GPU)");
}
