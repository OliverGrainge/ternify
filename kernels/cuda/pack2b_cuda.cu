#include "hip/hip_runtime.h"
#include "pack2b_cuda.h"
#include <hip/hip_runtime.h>

__global__ void _pack2b_kernel(const int8_t *d_A, int64_t M, int64_t N, int8_t *d_B)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N / 4)
    {
        int64_t base_idx = row * N + (col * 4);
        int8_t packed_value = 0;
        int64_t out_idx = row * (N / 4) + col;

        int8_t val0 = d_A[base_idx + 0] & 0x3;
        int8_t val1 = d_A[base_idx + 1] & 0x3;
        int8_t val2 = d_A[base_idx + 2] & 0x3;
        int8_t val3 = d_A[base_idx + 3] & 0x3;

        packed_value = (val0 << 6) | (val1 << 4) | (val2 << 2) | val3;
        d_B[out_idx] = packed_value;
    }
}

void _pack2b_cuda(const int8_t *d_A, int64_t M, int64_t N, int8_t *d_B)
{
    dim3 blockDim(16, 16);                                                                  // Configure block size
    dim3 gridDim((N / 4 + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y); // Configure grid size

    // Launch kernel
    _pack2b_kernel<<<gridDim, blockDim>>>(d_A, M, N, d_B);
}
