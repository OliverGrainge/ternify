#include "hip/hip_runtime.h"
#include "unpack2b_cuda.h"
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void _unpack2b_kernel(const int8_t *d_A, int64_t M, int64_t N, int8_t *d_B)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // N is the total number of columns after unpacking
    int64_t packed_cols = (N + 3) / 4;

    if (row < M && col < N)
    {
        // Calculate the index of the packed value
        int64_t b_idx = col >> 2;            // col / 4
        int64_t shift = (3 - (col & 3)) * 2; // (col % 4)

        // Each element in A_row contains 4 2-bit values
        const int8_t *A_row = d_A + row * packed_cols;

        // Unpack the 2-bit value
        d_B[row * N + col] = (A_row[b_idx] >> shift) & 0b11;
    }
}

void _unpack2b_cuda(const int8_t *d_A, int64_t M, int64_t N, int8_t *d_B)
{
    // Define CUDA thread block and grid dimensions
    dim3 blockDim(16, 16); // 16x16 threads per block
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    _unpack2b_kernel<<<gridDim, blockDim>>>(d_A, M, N, d_B);

    // Check for kernel errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
}
