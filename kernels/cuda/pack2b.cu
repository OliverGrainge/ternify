#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <cstdint>

__global__ void _pack2b_cuda_kernel(const int8_t *A, int64_t M, int64_t N, int8_t *B)
{
    int64_t row = blockIdx.y * blockDim.y + threadIdx.y;
    int64_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N / 4)
    {
        const int64_t base_idx = row * N + col * 4;
        int8_t packed_value = 0;

        const int8_t val0 = A[base_idx + 0] & 0x3;
        const int8_t val1 = A[base_idx + 1] & 0x3;
        const int8_t val2 = A[base_idx + 2] & 0x3;
        const int8_t val3 = A[base_idx + 3] & 0x3;

        packed_value = (val0 << 6) | (val1 << 4) | (val2 << 2) | val3;
        B[row * (N / 4) + col] = packed_value;
    }
}

torch::Tensor pack2b_cuda(torch::Tensor A)
{
    TORCH_CHECK(A.device().is_cuda(), "Tensor 'A' must be on CUDA device");
    TORCH_CHECK(A.dtype() == torch::kInt8, "Tensor 'A' must be of type int8");
    TORCH_CHECK(A.dim() == 2, "Tensor 'A' must be 2-dimensional");
    TORCH_CHECK(A.size(1) % 4 == 0, "Width must be a multiple of 4");

    A = A.contiguous(); // Ensure contiguous layout

    int64_t M = A.size(0);
    int64_t N = A.size(1);
    int64_t packed_cols = N / 4;

    auto C = torch::zeros({M, packed_cols}, torch::dtype(torch::kInt8).device(torch::kCUDA));

    dim3 blockDim(32, 32); // Adjust for optimal performance
    dim3 gridDim((packed_cols + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    _pack2b_cuda_kernel<<<gridDim, blockDim>>>(A.data_ptr<int8_t>(), M, N, C.data_ptr<int8_t>());

    hipDeviceSynchronize(); // Ensure kernel completion

    return C;
}